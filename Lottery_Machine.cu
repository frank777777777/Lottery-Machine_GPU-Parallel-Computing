// Simulator a lottery system
//
// Author: Yili Zou
// 
// For the GPU Programming class, NDSU Spring '14



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#define FILE_CREATE_ERROR -1
#define Number_Max 9
#define Number_Min 0


#define THREADS_PER_BLOCK 10 // Setting the grid up
#define BLOCKS_PER_GRID 1
#define OFFSET 0 // No offset

__global__ void Setup_RNG(hiprandState *state, int seed)
{
	// Setup of the random number generator. It seeds it, sets the sequence according to the thread id
	hiprand_init(seed, threadIdx.x + blockIdx.x * THREADS_PER_BLOCK, OFFSET, &state[threadIdx.x + blockIdx.x * THREADS_PER_BLOCK]);
	
}

__global__ void RNG(hiprandState *state,  int *result)
{
	int id_k = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK; // Here we calculate the id_k as to save calculations

	hiprandState localState = state[id_k]; // Copy it to local memory to save global memory accesses (faster)

	result[id_k] = hiprand(&localState)/(RAND_MAX/5); // Use the state to generate the random number AND updates the state,the range will be from 0 to 9, which is a dice

	state[id_k] = localState; // Update the state in global memory. This allows the next generation to be uncorrelated to this generation

}
__global__ void Number_Matching(int *lucky_numbers, int *user_numbers, int *Matching_numbers)
{
	//set up a counter to see how many numbers are matching
	int counter=0;  //initialize the counter, 0 is not matching, 1 is matching. 

	if(lucky_numbers[threadIdx.x]==user_numbers[threadIdx.x])
	{
		counter++;
	}

	Matching_numbers[threadIdx.x]= counter; //for every index that is matching, counter becomes 1, so this is a array of where these matching numbers are
	
}

int main()
{	
	//the array to store users number
	int user_number[10];
	//the array to store the lucky number
	int price_number[10];
	//define a address to store randomnumbers on the device
	int *randomnumbers_d;
	//how many numbers are matching
	int numbers_matching[10];
	//define stuff in device
	int *price_number_d;
	int *user_number_d;
	int *numbers_matching_d;
	
	//States
    hiprandState *states_d;
	
	// Allocate memory on the device
    hipMalloc((void **)&randomnumbers_d, THREADS_PER_BLOCK*sizeof( int)); 
	hipMalloc((void **)&states_d, THREADS_PER_BLOCK*sizeof(hiprandState));
    
	// Set up grid and block
	dim3 dimGrid(BLOCKS_PER_GRID);
	dim3 dimBlock(THREADS_PER_BLOCK); 
	
	// Set up RNG
	Setup_RNG<<<dimGrid, dimBlock>>>(states_d, time(NULL)); 

	RNG<<<dimGrid, dimBlock>>>(states_d, randomnumbers_d); // Launch RNG
	
	//copy results back
	hipMemcpy(price_number, randomnumbers_d, THREADS_PER_BLOCK*sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	//user interface
	printf("\nThe 10 lucky number have been generated, please input your lucky numbers!!(from 0 to 9)\n");
	//encourage user to input numbers
	int input; //the input by users
	for(int i=0; i<10; i++)
	{
		while(1)
		{
			scanf("%d", &input);  //scan the input
			if(input<0 || input >9)
			{
				printf("\n Please enter numbers within 0 to 9!\n"); //encourage to input valide number
			}
			else
			break;
		}
		user_number[i]=input;
	}
	printf("\nYour lucky numbers have been picked, waiting for results\n");
	// Allocate memory on the device
    hipMalloc((void **)&numbers_matching_d, 10*sizeof( int)); 
    hipMalloc((void **)&price_number_d, THREADS_PER_BLOCK*sizeof( int)); 
	hipMalloc((void **)&user_number_d, THREADS_PER_BLOCK*sizeof( int)); 
	
	//copy the parameters in the device
	hipMemcpy(user_number_d, user_number, THREADS_PER_BLOCK*sizeof(int ), hipMemcpyHostToDevice);
	hipMemcpy(price_number_d, price_number, THREADS_PER_BLOCK*sizeof(int ), hipMemcpyHostToDevice);
	
	//Launch number matching kernel
	Number_Matching<<<dimGrid, dimBlock>>>(price_number_d, user_number_d, numbers_matching_d);
	//copy the result
	hipMemcpy(numbers_matching, numbers_matching_d, 10*sizeof(int), hipMemcpyDeviceToHost);
	//clean up memory
	 hipFree(numbers_matching_d);
	 hipFree(price_number_d);
	 hipFree(user_number_d);
	 
	 //how many numbers matching
	 int nMatch=0;
	 //show result
	 
	 printf("lucky numbers:\n");
	 for(int i=0; i<10; i++)
	 {
		nMatch+=numbers_matching[i];   //numbers_matching[i] is going to be either 1 or 0, so sum them all up we can get the totally numbers matching
		printf("%d\n",price_number[i]);
	 }
	 printf("\n You have %d numbers matching!", nMatch);
	 if(nMatch==10) //when all matches, win the price, which is not likely to happen
	 {
		printf("\n Conflagrations! You have won 1 Million dollars! \n");
	 }
	 return EXIT_SUCCESS;
}
